#include "hip/hip_runtime.h"
__global__ void histo_kernel( uchar4 *buffer,
                               uint4 *partialHisto )
{
     __shared__ uint4 temp[256]; // 256*8 = 4096 = 4 blocks allowed
	  //256*3 channel = 3 blocks allowed
     temp[threadIdx.x] = 0; //?
     __syncthreads();

     int i = threadIdx.x + blockIdx.x * blockDim.x;
     int offset = i + blockDim.x * gridDim.x;
     //i += offset;
	 atomicAdd( &temp[buffer[offset].x].x, 1);
	 atomicAdd( &temp[buffer[offset].y].y, 1);
	 atomicAdd( &temp[buffer[offset].z].z, 1);
	 atomicAdd( &temp[buffer[offset].w].w, 1);
     
     __syncthreads();

    atomicAdd( &(histo[threadIdx.x]), temp[threadIdx.x] );
}

