
#include <hip/hip_runtime.h>
/*
 * from https://www.shadertoy.com/view/MdBGDK
 */

#define TAU 6.28318530718

#define _float2 make_float2
#define _float3 make_float3
#define _float4 make_float4

__device__ float mod(float x,float y) { return x - y * floorf(x/y); }
__device__ float dot(float2 v1, float2 v2) { return ( v1.x * v2.x + v1.y * v2.y ); }


/*__shared__ float gTime;
__shared__ float2 mou;*/
extern "C" __global__ 
void ribbon_assault (
        uchar4 *pixel,
        unsigned int width,
        unsigned int height,
        int time )
{
    // map from threadIdx/BlockIdx to pixel position
    float x = threadIdx.x + blockIdx.x * blockDim.x;
    float y = threadIdx.y + blockIdx.y * blockDim.y;
    uint offset = x + y * blockDim.x * gridDim.x;
    float W=width;
    float H=height;

    float gTime = time/1000.0f+11.0f;

    float f = 3.f, g = 3.f;

    //float2 mou = _float2(sin(gTime/10.0f)*W, cos(gTime/10.0f)*H);

    float2 mou = _float2(
                sin(gTime*.3)*sin(gTime*.17)*1.+sin(gTime*.3) ,
                (1.0-cos(gTime*.632f))*sin(gTime*.131f)*1.0f+cos(gTime*.3));
    mou.x=(mou.x+1.0f)*W;
    mou.y=(mou.y+1.0f)*H;


    float2 z = _float2((-W+2.0f*x)/H, (-H+2.0f*y)/H);
    float2 p = _float2((-W+2.0f+mou.x)/H, (-H+2.0f+mou.y)/H);


    for( int i = 0; i < 20; i++)
    {
        float d = dot(z,z);
        z = _float2( p.x+z.x/d, p.y/y-z.y/d );
        z.x =  abs(z.x);
        float2 ZP= _float2(z.x-p.x,z.y-p.y);
        float2 PZ= _float2(z.x+p.x,z.y+p.y);
        f = max( f, (dot(ZP,ZP) ));
        g = min( g, sin(mod(dot(PZ,PZ), TAU))+1.0f);
    }
    f = abs(-log(f) / 3.5f);
    g = abs(-log(g) / 8.0f);
    float3 V=_float3(min(g,1.0f), min(g*f,1.0f), min(f,1.0f) );

    pixel[offset].x = V.z*255;
    pixel[offset].y = V.y*255;
    pixel[offset].z = V.x*255;
    pixel[offset].w = 255;

}
