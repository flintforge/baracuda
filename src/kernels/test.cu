#include "hip/hip_runtime.h"
#include "baracuda2.hcu"

extern "C" __global__
void test (
        rgba8 * pixels,
        uint width,
        uint height,
        int time
        )
{
    _GLSL_SAUCE_;

    vec2 m=vec2(128,0);
    vec2 n = m;
    n.x = 0;
    n.y = 128;
    vec2 o = m + n;

    pixels[OFFSET].x = o.x;
    pixels[OFFSET].y = o.y;
    pixels[OFFSET].z = 128;
    pixels[OFFSET].w = 255;
}
