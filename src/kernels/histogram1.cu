#include "hip/hip_runtime.h"


// images are 16x16 chunks. dividing by 256 will is integral
// compute as many histograms as shared memory allows
// TODO : consider streaming with very big images.
extern "C" __global__ 
void subhistograms24 ( 
	uchar4 *img,
	uchar4 *hist,
	unsigned int width,
	unsigned int height
	) 
{
	__shared__ uchar4 subhist[256]; // 256*4 = 1024 => 16 blocs max
	float x = threadIdx.x + blockIdx.x * blockDim.x;
	subhist [ threadIdx.x ] = make_uchar4(0,0,0,0);
   __syncthreads(); 
	atomicAdd( &(subhist [ img[i].x ].x) , 1 );
	atomicAdd( &(subhist [ img[i].y ].y) , 1 );
	atomicAdd( &(subhist [ img[i].z ].z) , 1 );
	atomicAdd( &(subhist [ img[i].w ].w) , 1 );
	__syncthreads();
	atomicAdd( &(hist[threadIdx.x]), subhist[threadIdx.x] );
/* 
	hist[i].x = 255-img[i].x;
	hist[i].y = 255-img[i].y;
	hist[i].z = 255-img[i].z;
	 // */
}

