
#include <hip/hip_runtime.h>

////////////////////////////////////////////////////////////////////////////////
// GPU-specific common definitions
////////////////////////////////////////////////////////////////////////////////
#define uchar unsigned char
#define LOG2_WARP_SIZE 5U
#define WARP_SIZE (1U << LOG2_WARP_SIZE)
#define UINT_BITS 32
#define UCHAR_BITS 8
#define HISTO256_BINS 256
//Warps ==subhistograms per threadblock
#define WARP_COUNT 3
//Threadblock size
#define HISTOGRAM256_THREADBLOCK_SIZE (WARP_COUNT * WARP_SIZE)
//Shared memory per threadblock
#define HISTOGRAM256_THREADBLOCK_MEMORY (WARP_COUNT * HISTO256_BINS)

#define UMUL(a, b) ( (a) * (b) )
#define UMAD(a, b, c) ( UMUL((a), (b)) + (c) )

////////////////////////////////////////////////////////////////////////////
// Shortcut shared memory atomic addition functions
////////////////////////////////////////////////////////////////////////////////
#define USE_SMEM_ATOMICS 0

#if(!USE_SMEM_ATOMICS)
//#define TAG_MASK ( (1U << (UINT_BITS - LOG2_WARP_SIZE)) - 1U )
#define TAG_MASK ( (1U << (UCHAR_BITS - LOG2_WARP_SIZE)) - 1U )

inline __device__ void addByte(volatile uint *s_WarpHist, uchar data, uint threadTag)
{
    uint count;
    do
    {
        count = s_WarpHist[data] & TAG_MASK;
        count = threadTag | (count + 1);
        s_WarpHist[data] = count;
    }
    while (s_WarpHist[data] != count);
}
#else
#ifdef CUDA_NO_SM12_ATOMIC_INTRINSICS
#error Compilation target does not support shared-memory atomics
#endif

#define TAG_MASK 0xFFFFFFFFU
inline __device__ void addByte(uint4 *s_WarpHist, uint data, uint threadTag)
{
    atomicAdd(s_WarpHist + data, 1);
}
#endif

inline __device__ void addWord(uint *s_WarpHist, uchar data, uint tag)
{
    addByte(s_WarpHist, (data >>  0) & 0xFFU, tag);
    addByte(s_WarpHist, (data >>  8) & 0xFFU, tag);
    addByte(s_WarpHist, (data >> 16) & 0xFFU, tag);
    addByte(s_WarpHist, (data >> 24) & 0xFFU, tag);
}

extern "C"
__global__ void histogram256Kernel(uint *d_PartialHistograms, uchar4 *d_Data, uint dataCount)
{
    //Per-warp subhistogram storage
   __shared__ uint s_HistR[HISTOGRAM256_THREADBLOCK_MEMORY];
	__shared__ uint s_HistG[HISTOGRAM256_THREADBLOCK_MEMORY];
	__shared__ uint s_HistB[HISTOGRAM256_THREADBLOCK_MEMORY];
	__shared__ uint s_HistA[HISTOGRAM256_THREADBLOCK_MEMORY];
   uint *s_WarpHistR= s_HistR + (threadIdx.x >> LOG2_WARP_SIZE) * HISTO256_BINS;
	uint *s_WarpHistG= s_HistG + (threadIdx.x >> LOG2_WARP_SIZE) * HISTO256_BINS;
	uint *s_WarpHistB= s_HistB + (threadIdx.x >> LOG2_WARP_SIZE) * HISTO256_BINS;
	uint *s_WarpHistA= s_HistA + (threadIdx.x >> LOG2_WARP_SIZE) * HISTO256_BINS;

    //Clear shared memory storage for current threadblock before processing
#pragma unroll

    for (uint i = 0; i < (HISTOGRAM256_THREADBLOCK_MEMORY / HISTOGRAM256_THREADBLOCK_SIZE); i++)
    {
      s_HistR[threadIdx.x + i * HISTOGRAM256_THREADBLOCK_SIZE] = 0;
		s_HistG[threadIdx.x + i * HISTOGRAM256_THREADBLOCK_SIZE] = 0;
		s_HistB[threadIdx.x + i * HISTOGRAM256_THREADBLOCK_SIZE] = 0;
		s_HistA[threadIdx.x + i * HISTOGRAM256_THREADBLOCK_SIZE] = 0;
    }

    //Cycle through the entire data set, update subhistograms for each warp
    const uint tag = threadIdx.x << (UCHAR_BITS - LOG2_WARP_SIZE);

    __syncthreads();

    for (uint pos = UMAD(blockIdx.x, blockDim.x, threadIdx.x); pos < dataCount; pos += UMUL(blockDim.x, gridDim.x))
    {
        uchar dataR = d_Data[pos].x;
		uchar dataG = d_Data[pos].y;
		uchar dataB = d_Data[pos].z;
		uchar dataA = d_Data[pos].w;
        addWord(s_WarpHistR, dataR, tag);
		addWord(s_WarpHistG, dataG, tag);
		addWord(s_WarpHistB, dataB, tag);
		addWord(s_WarpHistA, dataA, tag);
    }

    //Merge per-warp histograms into per-block and write to global memory
    __syncthreads();

    for (uint bin = threadIdx.x; bin < HISTO256_BINS; bin += HISTOGRAM256_THREADBLOCK_SIZE)
    {
        uint sumR = 0;
		uint sumG = 0;
		uint sumB = 0;
		uint sumA = 0;

        for (uint i = 0; i < WARP_COUNT; i++)
        {
			sumR += s_HistR[bin + i * HISTO256_BINS] & TAG_MASK;
			sumG += s_HistG[bin + i * HISTO256_BINS] & TAG_MASK;
			sumB += s_HistB[bin + i * HISTO256_BINS] & TAG_MASK;
			sumA += s_HistA[bin + i * HISTO256_BINS] & TAG_MASK;
        }

        d_PartialHistograms[blockIdx.x * HISTO256_BINS + bin] = sumR;
		d_PartialHistograms[blockIdx.x * HISTO256_BINS + bin] = sumG;
		d_PartialHistograms[blockIdx.x * HISTO256_BINS + bin] = sumB;
		d_PartialHistograms[blockIdx.x * HISTO256_BINS + bin] = sumA;
    }
}
