
#include <hip/hip_runtime.h>

extern "C" __global__ 
void invert ( uchar4 *frame ) 
{
    float x = threadIdx.x + blockIdx.x * blockDim.x;
    float y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    frame[offset].x = 255-frame[offset].x;
    frame[offset].y = 255-frame[offset].y; 
    frame[offset].z = 255-frame[offset].z;
}

