
#include <hip/hip_runtime.h>
/*
this reveal the block dimension
as adressed on the device.
blocks are according to the moduli
computed from the window size and the block size.

Change the window size to see it in effect
*/

extern "C" __global__ 
void dimblock (
        uchar4* ptr,
        unsigned int width,
        unsigned int height,
        int time
        )
{
    uint x = threadIdx.x + blockIdx.x * blockDim.x;
    uint y = threadIdx.y + blockIdx.y * blockDim.y;
    uint offset = x + y * blockDim.x * gridDim.x;

    float t= sin(time/1000.0f);
    float X = (float)x/(float)width;
    float Y = (float)y/(float)height;
    ptr[offset].x = (unsigned char) ( (sinf( X + t ) + threadIdx.x ) * 255 );
    ptr[offset].y = (unsigned char) ( (sinf( Y + t ) + threadIdx.y ) * 255 );
    ptr[offset].z = sin((float)time/1000.0f ) *255;
    ptr[offset].w = 255;

}

