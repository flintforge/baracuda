#include "hip/hip_runtime.h"
#include "baracuda.hcu"

extern "C" __global__
void aliasingCheck2 (
        rgba8 * pixels,
        uint width,
        uint height,
        int time
        )
{
    _GLSL_SAUCE_;

    float stime = time/1000.0f;

    /* vec2 RG = vec2(FragCoord.xy);

    vec3 V = vec3( 1, 1, sin(gTime) );
                //RG.x,RG.y);

    vec4 P= vec4(V,1.f);
    V*=255.0f;
    pixels << V;*/

    float sX = ((float)_px_-(float)width/2)/(float)width;
    float sY = ((float)_py_-(float)height/2)/(float)height;
    int D = 255.0f*abs(sqrtf((sX+1.0f)*sX+sY*sY)-stime);

    vec3 px = D;

    pixels[OFFSET].x = px.x ;
    pixels[OFFSET].y = D ;
    pixels[OFFSET].z = D ;
    pixels[OFFSET].w = 255;

}

