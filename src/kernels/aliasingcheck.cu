#include "hip/hip_runtime.h"
#include "baracuda.hcu"

extern "C" __global__
void aliasingcheck (
        rgba8 * pixels,
        uint width,
        uint height,
        int time
        )
{
    _GLSL_SAUCE_;

    float D = abs(sqrtf(
                      (1.f+FragCoord.x) * FragCoord.x
                      + FragCoord.y*FragCoord.y
                      )-time/10.0f);

    vec3 px = D;

    /*
    this crash at src/op.cpp>, line 114. hipCtxSynchronize
    vec3 R;
    R = px; // pb passing ref ptr ? vec3(px) is ok.
    pixels << R;
    // */
    Output3(pixels, px);

}
