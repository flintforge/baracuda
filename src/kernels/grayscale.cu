
#include <hip/hip_runtime.h>


// turn 
extern "C" __global__ 
void grayscale ( uchar4 *img ) 
{
    float x = threadIdx.x + blockIdx.x * blockDim.x;
    float y = threadIdx.y + blockIdx.y * blockDim.y;
    int i = x + y * blockDim.x * gridDim.x;
    unsigned char mean = (img[i].x + img[i].y + img[i].z)/3;
    img[i].x = img[i].y = img[i].z = mean;
}

