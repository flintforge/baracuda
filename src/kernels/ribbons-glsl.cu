#include "hip/hip_runtime.h"

#include "baracuda.hcu"
#define TAU 6.28318530718

extern "C" __global__
void ribbon_glsl(
        uchar4 *frame,
        unsigned int width,
        unsigned int height,
        int time
         )
{
    _GLSL_SAUCE_

    float gTime = time/1000.0f+11.0f;

    float f = 3.f, g = 3.f;

    vec2 mu;
    mu.x = sin(gTime*.3) * sin(gTime*.17)*1.f + sin(gTime*.3);
    mu.y = (1.0-cos(gTime*.632f)) * sin(gTime*.131f)*1.f + cos(gTime*.3);
    mu = (mu+1) * FragSize;

    vec2 z = ((-FragSize + FragCoord.xy*2.0f) / FragSize.y );
    vec2 p = ((-FragSize + 2.0f + mu ) / FragSize.y );


    for( int i = 0; i < 20; i++)
    {
            float d = dot(z,z);
            z = (vec2(z.x,-z.y ) / d)+p;
            z.x =  abs(z.x);
            f = max( f, (dot(z-p,z-p) ));
            g = min( g, sin(mod(dot(z+p,z+p), TAU))+1.0f);
    }
    f = abs(-log(f) / 3.5f);
    g = abs(-log(g) / 8.0f);
    vec3 V = min(vec3(f,f*g,g),1.0f);

    V *=255;
    Output3(frame,V);
}
