
#include <hip/hip_runtime.h>


extern "C" 
__global__ 
void mandelbrot ( 
	uchar4 *ptr, 
	unsigned int width, 
	unsigned int height, 
	int time) 
{


	int iter=48;
	// map from threadIdx/BlockIdx to pixel position
    float x = threadIdx.x + blockIdx.x * blockDim.x;
    float y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    
	  float cx = 3.0f * (x / width-0.5f);
	  float cy = 2.0f * (y / height-0.5f);

	  int i;
    float zx = cx;
    float zy = cy;
		float dx = 0.0f;
		float dy = 0.0f; 
    cx *=  cos(time/1000.0f);
    cy *=  sin(time/1000.0f);
	  for(i=0; i<iter; i++) {
      dx = (zx * zx - zy * zy) + cx + dx;
      dy = (zy * zx + zx * zy) + cy + dy;
		  if((dx * dx + dy * dy) > 4.0f) break;
		  zx = dx;
		  zy = dy;
	  }

	  unsigned char val = 255.0f*float(i==iter?0:i)/iter;
    
    ptr[offset].x = min(255,3*val/2);
    ptr[offset].y = (val>100 ? min(255,3*(val-100)) : 0);
    ptr[offset].z = (val>127 ? min(255,2*(val-127)) : 0);
    ptr[offset].w = 255;
}

